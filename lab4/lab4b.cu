
#include <hip/hip_runtime.h>
__global__
void f1( float4* __restrict__ ptr ) {
  float4 v = ptr[threadIdx.x];
  v.x += 1;
  v.y += 1;
  v.z += 1;
  v.w += 1;
  ptr[threadIdx.x] = v;
}

__global__
void f2( float* __restrict__ ptr1, float* __restrict__ ptr2, float* __restrict__ ptr3, float* __restrict__ ptr4 ) {
  ptr1[threadIdx.x] += 1;
  ptr2[threadIdx.x] += 1;
  ptr3[threadIdx.x] += 1;
  ptr4[threadIdx.x] += 1;
}

int main() {
  float *some_ptr;
  hipMalloc(&some_ptr, 128 * sizeof(float));
  f1<<<1, 32>>>((float4*) some_ptr);
  f2<<<1, 32>>>(some_ptr, some_ptr+32, some_ptr+64, some_ptr+96);
}
